#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <stdio.h>
#include <cmath>

int INF = 100;


template <typename scalar_t>
__global__ void maxpool_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int STRIDE, const int K, const int W_OUT, const int PLANE) {
  // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h_in = (cell_idx / W_OUT) * STRIDE;
  const int w_in = fmod(cell_idx, W_OUT) * STRIDE;
  const int h_out = (cell_idx / W_OUT);
  const int w_out = fmod(cell_idx, W_OUT);
  // If this provenance has already been recorded (and thus the value has been placed back), skip.
  if (provenance[b][c][h_out][w_out] > -1) return;
  // Keep temporary storage for output and provenance.
  scalar_t out_ = -100.0;
  int max_p_ = -1;
  int idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      if (input[b][c][h_in + i][w_in + j] > out_) {
        out_ = input[b][c][h_in + i][w_in + j];
        max_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  output[b][c][h_out][w_out] = out_;
  provenance[b][c][h_out][w_out] = max_p_;
}

std::vector<torch::Tensor> maxpool_forward_cuda(
  torch::Tensor input, const int kernel_size, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  // Pad the input with a large minus value if the kernel has odd size.
  const int pad = kernel_size / 2;
  if (kernel_size % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, -INF);
  }
  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));
  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "maxpool_forward_cuda", ([&] {
    maxpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, stride, kernel_size, W_OUT, PLANE_SIZE);
  }));
  hipDeviceSynchronize();
  return {output, provenance};
}

std::vector<torch::Tensor> maxpool_double_forward_cuda(
  torch::Tensor input, const int kernel_size, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  // Pad the input with a large minus value if the kernel has odd size.
  const int pad = kernel_size / 2;
  if (kernel_size % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, -INF);
  }
  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));
  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "maxpool_double_forward_cuda", ([&] {
    maxpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, stride, kernel_size, W_OUT, PLANE_SIZE);
  }));

  // In this function, we run the dilation twice to get rid of -1s.
  torch::Tensor copied_output;
  if (kernel_size % 2 == 0) {
    copied_output = torch::constant_pad_nd(output, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    copied_output = torch::constant_pad_nd(output, torch::IntList{pad, pad, pad, pad}, -INF);
  }

  AT_DISPATCH_ALL_TYPES(output.scalar_type(), "maxpool_double_forward_cuda", ([&] {
    maxpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        copied_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, 1, kernel_size, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}

template <typename scalar_t>
__global__ void parameterized_maxpool_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    unsigned int* poi_counter, const int THREADS, const int STRIDE, const int K, const int W_OUT, const int PLANE) {
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h_in = (cell_idx / W_OUT) * STRIDE;
  const int w_in = fmod(cell_idx, W_OUT) * STRIDE;
  const int h_out = (cell_idx / W_OUT);
  const int w_out = fmod(cell_idx, W_OUT);
  // If this provenance has already been recorded (and thus the value has been placed back), skip.
  if (provenance[b][c][h_out][w_out] > -1) return;

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();
  // Keep temporary storage for output and provenance.
  scalar_t out_ = -100.0;
  int max_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In dilation, the kernel is added from the patch.
      scalar_t val = input[b][c][h_in + i][w_in + j] + kdata[idx];
      if (val > out_) {
        out_ = val;
        max_p_ = idx;
      }
      idx++;
    }
  }
  
  // int poc_counter = 0;
  // idx = 0;
  // // Loop over to find number of PoC.
  // for (int i=0; i < K; i++) {
  //   for (int j=0; j < K; j++) {
  //     // In dilation, the kernel is added from the patch.
  //     scalar_t val = input[b][c][h_in + i][w_in + j] + kdata[idx];
  //     if (abs(val - out_) < 1e-9) {
  //       poc_counter += 1;
  //     }
  //     idx++;
  //   }
  // }

  // if (poc_counter >= 2) {
  //   atomicAdd(poi_counter, 1);
  //   // printf("Found point with multiple PoC: %d.\n", poc_counter);
  // }

  // Now assign to the correct output position and register provenance.
  output[b][c][h_out][w_out] = out_;
  provenance[b][c][h_out][w_out] = max_p_;
}

std::vector<torch::Tensor> parameterized_maxpool_forward_cuda(
  torch::Tensor input, torch::Tensor weights, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  const int K = weights.size(2);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  const int pad = K / 2;
  if (K % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, -INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));

  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  unsigned int poi_counter = 0;
  unsigned int* poi_counter_ptr;
  hipMalloc(&poi_counter_ptr, sizeof(unsigned int));
  hipMemcpy(poi_counter_ptr, &poi_counter, sizeof(unsigned int), hipMemcpyHostToDevice);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "parameterized_maxpool_forward_cuda", ([&] {
    parameterized_maxpool_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        poi_counter_ptr, threads, stride, K, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();

  hipMemcpy(&poi_counter, poi_counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost);
  hipFree(poi_counter_ptr);

  torch::Tensor poi_tensor = torch::tensor(torch::IntList{poi_counter}, torch::dtype(torch::kI32));
  
  // printf("POI Counter: %u.\n", poi_counter);
  return {output, provenance, poi_tensor};
}

template <typename scalar_t>
__global__ void parameterized_maxpool_first_of_two_passes_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int K, const int W_OUT, const int PLANE) {
  /*
    Specifically for parameterized double passes, we need two passes.
    The first pass imoutes the max values (with SE additions), but only puts
    the value at the provenance.
    In the second pass, all SE values are added to all locations.
  */
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h = (cell_idx / W_OUT);
  const int w = fmod(cell_idx, W_OUT);

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();
  // Keep temporary storage for output and provenance.
  scalar_t out_ = -100.0;
  scalar_t max_value_ = -100.0;
  int max_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In dilation, the kernel is added from the patch.
      scalar_t val = input[b][c][h + i][w + j] + kdata[idx];
      if (val > max_value_) {
        max_value_ = val;
        out_ = input[b][c][h + i][w + j];
        max_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  if (out_ > -90.) {
    output[b][c][h][w] = out_;
    provenance[b][c][h][w] = max_p_;
    return;
  }
  // However, if the element had no -inf around it, impute a very low value.
  output[b][c][h][w] = -100.;
  provenance[b][c][h][w] = -1;
}

template <typename scalar_t>
__global__ void parameterized_maxpool_second_of_two_passes_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int K, const int W_OUT, const int PLANE) {
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h = (cell_idx / W_OUT);
  const int w = fmod(cell_idx, W_OUT);
  const int pad = K / 2;

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();

  // If this provenance has already been recorded, add the kernel value
  // and use that as output.
  if (provenance[b][c][h][w] > -1) {
    int idx = provenance[b][c][h][w];
    output[b][c][h][w] = input[b][c][h + pad][w + pad] + kdata[idx];
    return;
  }

  // Keep temporary storage for output and provenance.
  scalar_t out_ = -100.0;
  int max_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In dilation, the kernel is added from the patch.
      scalar_t val = input[b][c][h + i][w + j] + kdata[idx];
      if (val > out_) {
        out_ = val;
        max_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  output[b][c][h][w] = out_;
  provenance[b][c][h][w] = max_p_;
}

std::vector<torch::Tensor> parameterized_maxpool_double_forward_cuda(
  torch::Tensor input, torch::Tensor weights, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  const int K = weights.size(2);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  const int pad = K / 2;
  if (K % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, -INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));

  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "parameterized_maxpool_double_forward_cuda", ([&] {
    parameterized_maxpool_first_of_two_passes_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, K, W_OUT, PLANE_SIZE);
  }));

  // In this function, we run the dilation twice to get rid of -1s.
  torch::Tensor copied_output;
  if (K % 2 == 0) {
    copied_output = torch::constant_pad_nd(output, torch::IntList{0, pad, 0, pad}, -INF);
  } else {
    copied_output = torch::constant_pad_nd(output, torch::IntList{pad, pad, pad, pad}, -INF);
  }

  AT_DISPATCH_FLOATING_TYPES(output.scalar_type(), "parameterized_maxpool_double_forward_cuda", ([&] {
    parameterized_maxpool_second_of_two_passes_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        copied_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, K, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}

template <typename scalar_t>
__global__ void minpool_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int STRIDE, const int K, const int W_OUT, const int PLANE) {
  // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h_in = (cell_idx / W_OUT) * STRIDE;
  const int w_in = fmod(cell_idx, W_OUT) * STRIDE;
  const int h_out = (cell_idx / W_OUT);
  const int w_out = fmod(cell_idx, W_OUT);
  // If this provenance has already been recorded (and thus the value has been placed back), skip.
  if (provenance[b][c][h_out][w_out] > -1) return;
  // Keep temporary storage for output and provenance.
  scalar_t out_ = 100.0;
  int min_p_ = -1;
  int idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      if (input[b][c][h_in + i][w_in + j] < out_) {
        out_ = input[b][c][h_in + i][w_in + j];
        min_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  output[b][c][h_out][w_out] = out_;
  provenance[b][c][h_out][w_out] = min_p_;
}

std::vector<torch::Tensor> minpool_forward_cuda(
  torch::Tensor input, const int kernel_size, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  // Pad the input with a large minus value if the kernel has odd size.
  const int pad = kernel_size / 2;
  if (kernel_size % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));
  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "minpool_forward_cuda", ([&] {
    minpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, stride, kernel_size, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}

std::vector<torch::Tensor> minpool_double_forward_cuda(
  torch::Tensor input, const int kernel_size, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  // Pad the input with a large minus value if the kernel has odd size.
  const int pad = kernel_size / 2;
  if (kernel_size % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));
  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "minpool_double_forward_cuda", ([&] {
    minpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, stride, kernel_size, W_OUT, PLANE_SIZE);
  }));

  // In this function, we run the dilation twice to get rid of -1s.
  torch::Tensor copied_output;
  if (kernel_size % 2 == 0) {
    copied_output = torch::constant_pad_nd(output, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    copied_output = torch::constant_pad_nd(output, torch::IntList{pad, pad, pad, pad}, INF);
  }

  AT_DISPATCH_ALL_TYPES(output.scalar_type(), "minpool_double_forward_cuda", ([&] {
    minpool_forward_kernel<scalar_t><<<blocks, threads>>>(
        copied_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, 1, kernel_size, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}

template <typename scalar_t>
__global__ void parameterized_minpool_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int STRIDE, const int K, const int W_OUT, const int PLANE) {
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h_in = (cell_idx / W_OUT) * STRIDE;
  const int w_in = fmod(cell_idx, W_OUT) * STRIDE;
  const int h_out = (cell_idx / W_OUT);
  const int w_out = fmod(cell_idx, W_OUT);
  // If this provenance has already been recorded (and thus the value has been placed back), skip.
  if (provenance[b][c][h_out][w_out] > -1) return;

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();
  // Keep temporary storage for output and provenance.
  scalar_t out_ = 100.0;
  int min_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In erosion, the kernel is subtracted from the patch.
      scalar_t val = input[b][c][h_in + i][w_in + j] - kdata[idx];
      if (val < out_) {
        out_ = val;
        min_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  output[b][c][h_out][w_out] = out_;
  provenance[b][c][h_out][w_out] = min_p_;
}

std::vector<torch::Tensor> parameterized_minpool_forward_cuda(
  torch::Tensor input, torch::Tensor weights, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  const int K = weights.size(2);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  const int pad = K / 2;
  if (K % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));

  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_ALL_TYPES(input.scalar_type(), "parameterized_minpool_forward_cuda", ([&] {
    parameterized_minpool_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, stride, K, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}

template <typename scalar_t>
__global__ void parameterized_minpool_first_of_two_passes_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int K, const int W_OUT, const int PLANE) {
  /*
    Specifically for parameterized double passes, we need two passes.
    The first pass imoutes the max values (with SE additions), but only puts
    the value at the provenance.
    In the second pass, all SE values are added to all locations.
  */
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h = (cell_idx / W_OUT);
  const int w = fmod(cell_idx, W_OUT);

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();
  // Keep temporary storage for output and provenance.
  scalar_t out_ = 100.0;
  scalar_t min_value_ = 100.0;
  int min_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In dilation, the kernel is added from the patch.
      scalar_t val = input[b][c][h + i][w + j] - kdata[idx];
      if (val < min_value_) {
        min_value_ = val;
        out_ = input[b][c][h + i][w + j];
        min_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  if (out_ < 90.) {
    output[b][c][h][w] = out_;
    provenance[b][c][h][w] = min_p_;
    return;
  }
  // However, if the element had no -inf around it, impute a very low value.
  output[b][c][h][w] = 100.;
  provenance[b][c][h][w] = -1;
}

template <typename scalar_t>
__global__ void parameterized_minpool_second_of_two_passes_forward_kernel(
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weights,
    torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> output,
    torch::PackedTensorAccessor32<int16_t,4,torch::RestrictPtrTraits> provenance,
    const int THREADS, const int K, const int W_OUT, const int PLANE) {
  // // batch and c_out indicex.
  const int b = blockIdx.x;
  const int c = blockIdx.y;
  // height and width indices.
  const int cell_idx = blockIdx.z * THREADS + threadIdx.x;
  if (cell_idx >= PLANE) return;
  const int h = (cell_idx / W_OUT);
  const int w = fmod(cell_idx, W_OUT);
  const int pad = K / 2;

  extern __shared__ unsigned char kdata_uchar[];
  scalar_t *kdata = reinterpret_cast<scalar_t *>(kdata_uchar);
  // Fill the shared memory.
  int idx = 0;
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      kdata[idx] = weights[c][i][j];
      idx++;
    }
  }
  __syncthreads();

  // If this provenance has already been recorded, add the kernel value
  // and use that as output.
  if (provenance[b][c][h][w] > -1) {
    int idx = provenance[b][c][h][w];
    output[b][c][h][w] = input[b][c][h + pad][w + pad] - kdata[idx];
    return;
  }

  // Keep temporary storage for output and provenance.
  scalar_t out_ = 100.0;
  int min_p_ = -1;
  idx = 0;
  // Loop over a kernel for a single pixel.
  for (int i=0; i < K; i++) {
    for (int j=0; j < K; j++) {
      // In dilation, the kernel is added from the patch.
      scalar_t val = input[b][c][h + i][w + j] - kdata[idx];
      if (val < out_) {
        out_ = val;
        min_p_ = idx;
      }
      idx++;
    }
  }
  // Now assign to the correct output position and register provenance.
  output[b][c][h][w] = out_;
  provenance[b][c][h][w] = min_p_;
}

std::vector<torch::Tensor> parameterized_minpool_double_forward_cuda(
  torch::Tensor input, torch::Tensor weights, const int stride, const int device) {
  // Get the dimensions of the operation.
  const int B = input.size(0);
  const int C = input.size(1);
  const int H_IN = input.size(2);
  const int W_IN = input.size(3);
  const int K = weights.size(2);
  // We need the output sizes of the operation. PyTorch has a way of cutting
  // off edges of the image if the sizes are odd. This is to replicate that behaviour.
  const int H_OUT = (H_IN + (stride > 1 ? 1 : 0)) / stride;
  const int W_OUT = (W_IN + (stride > 1 ? 1 : 0)) / stride;
  const int PLANE_SIZE = H_OUT * W_OUT;
  const int pad = K / 2;
  if (K % 2 == 0) {
    input = torch::constant_pad_nd(input, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    input = torch::constant_pad_nd(input, torch::IntList{pad, pad, pad, pad}, INF);
  }

  // Initialize the output volume, compensated for the amount of stride.
  torch::Tensor output = torch::empty(torch::IntList{B, C, H_OUT, W_OUT}, torch::dtype(torch::kF32).device(torch::kCUDA, device));
  // Initialize a volume to track the provenance of maximum values.
  torch::Tensor provenance = torch::full(torch::IntList{B, C, H_OUT, W_OUT}, -1, torch::dtype(torch::kI16).device(torch::kCUDA, device));

  const int threads = 192;
  const int Z = (H_OUT * W_OUT + threads - 1) / threads;
  const dim3 blocks(B, C, Z);

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "parameterized_minpool_double_forward_cuda", ([&] {
    parameterized_minpool_first_of_two_passes_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        input.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, K, W_OUT, PLANE_SIZE);
  }));

  // In this function, we run the dilation twice to get rid of -1s.
  torch::Tensor copied_output;
  if (K % 2 == 0) {
    copied_output = torch::constant_pad_nd(output, torch::IntList{0, pad, 0, pad}, INF);
  } else {
    copied_output = torch::constant_pad_nd(output, torch::IntList{pad, pad, pad, pad}, INF);
  }

  AT_DISPATCH_ALL_TYPES(output.scalar_type(), "parameterized_minpool_double_forward_cuda", ([&] {
    parameterized_minpool_second_of_two_passes_forward_kernel<scalar_t><<<blocks, threads, K*K*sizeof(scalar_t)>>>(
        copied_output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        weights.packed_accessor32<scalar_t,3,torch::RestrictPtrTraits>(),
        output.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
        provenance.packed_accessor32<int16_t,4,torch::RestrictPtrTraits>(),
        threads, K, W_OUT, PLANE_SIZE);
  }));

  hipDeviceSynchronize();
  return {output, provenance};
}
